#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "sobel.h"

#define GRIDVAL 20.0 
#define BLOCK_WIDTH 32
#define PI 3.14159265

void sobel(int width, int height, int channels, unsigned char *image)
{    
    int img2d[height][width];

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            img2d[i][j] = image[i * width + j];
        }
    }
    int img2dhororg[height][width];
    int img2dverorg[height][width];
    int img2dmag[height][width];

    ///horizontal
    int max = -200, min = 2000;

    for (int i = 1; i < height - 1; i++)
    {
        for (int j = 1; j < width - 1; j++)
        {
            int curr = img2d[i - 1][j - 1] + 2 * img2d[i - 1][j] + img2d[i - 1][j + 1] - img2d[i + 1][j - 1] - 2 * img2d[i + 1][j] - img2d[i + 1][j + 1];
            img2dhororg[i][j] = curr;
            if (curr > max)
                max = curr;
            if (curr < min)
                min = curr;
        }
    }

    ///vertical:
    max = -200;
    min = 2000;

    for (int i = 1; i < height - 1; i++)
    {
        for (int j = 1; j < width - 1; j++)
        {
            int curr = img2d[i - 1][j - 1] + 2 * img2d[i][j - 1] + img2d[i + 1][j - 1] - img2d[i - 1][j + 1] - 2 * img2d[i][j + 1] - img2d[i + 1][j + 1];
            img2dverorg[i][j] = curr;
            if (curr > max)
                max = curr;
            if (curr < min)
                min = curr;
        }
    }

    ///magnitude
    max = -200;
    min = 2000;

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            img2dmag[i][j] = sqrt(pow(img2dhororg[i][j], 2) + pow(img2dverorg[i][j], 2));
            if (img2dmag[i][j] > max)
                max = img2dmag[i][j];
            if (img2dmag[i][j] < min)
                min = img2dmag[i][j];
        }
    }

    int diff = max - min;

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            float abc = (img2dmag[i][j] - min) / (diff * 1.0);
            img2dmag[i][j] = abc * 255;
        }
    }

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            image[i * width + j] = img2dmag[i][j];
        }
    }
}  

/************************************************************************************************
* void sobel_gpu(const byte*, byte*, uint, uint);
* - This function runs on the GPU, it works on a 2D grid giving the current x, y pair being worked
* - on, the const byte* is the original image being processed and the second byte* is the image
* - being created using the sobel filter. This function runs through a given x, y pair and uses 
* - a sobel filter to find whether or not the current pixel is an edge, the more of an edge it is
* - the higher the value returned will be
* 
* Inputs: const byte* orig : the original image being evaluated
*                byte* cpu : the image being created using the sobel filter
*               uint width : the width of the image
*              uint height : the height of the image
* 
***********************************************************************************************/
__global__ void sobel_gpu(
    const unsigned char* orig, 
    unsigned char* cpu, 
    const unsigned int width, 
    const unsigned int height) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
                (    orig[(y-1)*width + (x+1)]) + ( 2*orig[y*width+(x+1)]) + (   orig[(y+1)*width+(x+1)]);
        dy = (    orig[(y-1)*width + (x-1)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+1)]) +
                (-1* orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
        cpu[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
    }
}

void sobel_optimized(int width, int height, int channels, unsigned char *image)
{
    /** Finally, we use the GPU to parallelize it further **/
    /** Allocate space in the GPU for our original img, new img, and dimensions **/
    unsigned char *gpu_orig, *gpu_sobel;
    hipMalloc( (void**)&gpu_orig, (width * height));
    hipMalloc( (void**)&gpu_sobel, (width * height));
    /** Transfer over the memory from host to device and memset the sobel array to 0s **/
    hipMemcpy(gpu_orig, image, (width * height), hipMemcpyHostToDevice);
    hipMemset(gpu_sobel, 0, (width * height));

    /** set up the dim3's for the gpu to use as arguments (threads per block & num of blocks)**/
    dim3 threadsPerBlock(GRIDVAL, GRIDVAL, 1);
    dim3 numBlocks(ceil(width/GRIDVAL), ceil(height/GRIDVAL), 1);

    /** Run the sobel filter using the CPU **/
    //  c = std::chrono::system_clock::now();
    sobel_gpu<<<numBlocks, threadsPerBlock>>>(gpu_orig, gpu_sobel, width, height);
    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if ( cudaerror != hipSuccess ) fprintf( stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName( cudaerror ) ); // if error, output error
    //  std::chrono::duration<double> time_gpu = std::chrono::system_clock::now() - c;
    /** Copy data back to CPU from GPU **/
    hipMemcpy(image, gpu_sobel, (width*height), hipMemcpyDeviceToHost);


    /** Free any memory leftover.. gpuImig, cpuImg, and ompImg get their pixels free'd while writing **/
    hipFree(gpu_orig); hipFree(gpu_sobel);
}  
