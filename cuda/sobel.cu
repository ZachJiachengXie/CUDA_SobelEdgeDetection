#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <jpeglib.h>
#include <iostream>
#include <cmath>
#include <fstream>

using namespace std;
#define PI 3.14159265

#include "sobel.h"

void sobel(int width, int height, int channels, unsigned char *image)
{
    int img2d[height][width];

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            img2d[i][j] = image[i * width + j];
        }
    }
    int img2dhororg[height][width];
    int img2dverorg[height][width];
    int img2dmag[height][width];

    ///horizontal
    int max = -200, min = 2000;

    for (int i = 1; i < height - 1; i++)
    {
        for (int j = 1; j < width - 1; j++)
        {
            int curr = img2d[i - 1][j - 1] + 2 * img2d[i - 1][j] + img2d[i - 1][j + 1] - img2d[i + 1][j - 1] - 2 * img2d[i + 1][j] - img2d[i + 1][j + 1];
            img2dhororg[i][j] = curr;
            if (curr > max)
                max = curr;
            if (curr < min)
                min = curr;
        }
    }

    ///vertical:
    max = -200;
    min = 2000;

    for (int i = 1; i < height - 1; i++)
    {
        for (int j = 1; j < width - 1; j++)
        {
            int curr = img2d[i - 1][j - 1] + 2 * img2d[i][j - 1] + img2d[i + 1][j - 1] - img2d[i - 1][j + 1] - 2 * img2d[i][j + 1] - img2d[i + 1][j + 1];
            img2dverorg[i][j] = curr;
            if (curr > max)
                max = curr;
            if (curr < min)
                min = curr;
        }
    }

    ///magnitude
    max = -200;
    min = 2000;

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            img2dmag[i][j] = sqrt(pow(img2dhororg[i][j], 2) + pow(img2dverorg[i][j], 2));
            if (img2dmag[i][j] > max)
                max = img2dmag[i][j];
            if (img2dmag[i][j] < min)
                min = img2dmag[i][j];
        }
    }

    int diff = max - min;

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            float abc = (img2dmag[i][j] - min) / (diff * 1.0);
            img2dmag[i][j] = abc * 255;
        }
    }

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            image[i * width + j] = img2dmag[i][j];
        }
    }
}  
